#include "hip/hip_runtime.h"
#include "ATen/ops/empty_like.h"
#include "kernel.cuh"
#include "utils.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <torch/nn/functional.h>

void layer_norm(float *input, float *output, float *gamma, float *beta, int N,
                int feature_size);

inline std::pair<float, float> compute_sol(float duration_ms,
                                           size_t input_size) {
  float compute_sol = input_size / kA100TOPS / 1E9;
  float memory_sol = float(input_size) * sizeof(float) / kA100MemBandwidth / 1E6;
  return {compute_sol / duration_ms,
          std::max<float>(compute_sol, memory_sol) / duration_ms}; // 'mfu', sol
}

int main() {
  int n = 10000, f = 2048;
  float gamma = 1, beta = 0;

  auto input = at::randn({n, f}, at::kCUDA);
  auto output_torch = torch::nn::functional::layer_norm(
      input, torch::nn::functional::LayerNormFuncOptions({f}));
  auto profiler = Profiler();

  auto input_ptr = reinterpret_cast<float *>(input.data_ptr());
  auto output_cuda = at::empty_like(output_torch);
  auto output_ptr = reinterpret_cast<float *>(output_cuda.data_ptr());

  float total_time = 0;

  int warmup = 10, repeat = 10;
  // warmup
  for (int i = 0; i < warmup; i++) {
    layer_norm(input_ptr, output_ptr, &gamma, &beta, n, f);
  }

  // run kernel
  for (int i = 0; i < repeat; i++) {
    profiler.start();
    layer_norm(input_ptr, output_ptr, &gamma, &beta, n, f);
    total_time += profiler.get_elasped(); // ms
  }
  float avg_time = total_time / repeat;

  // check result
  auto result = Result();
  result.bound_type = BoundType::Memory;
  result.is_correct = at::allclose(output_cuda, output_torch, 0.001, 0.001);
  auto mfu_sol = compute_sol(avg_time, n * f + 2 * f);

  result.sol = mfu_sol.second;
  result.is_too_slow = result.sol < 0.3;

  std::cout << result;
  return 0;
}
