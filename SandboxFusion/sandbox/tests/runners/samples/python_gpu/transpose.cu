#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void forward_kernel(const float *input, float *output, int M, int N) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    output[x * M + y] = input[y * N + x];
}

torch::Tensor forward(torch::Tensor input) {
    assert(input.ndimension() == 2);
    const int64_t M = input.size(0);
    const int64_t N = input.size(1);
    std::vector<int64_t> output_shape{N, M};
    torch::Tensor output = torch::empty(output_shape, input.options());

    dim3 threads(32, 32);
    dim3 blocks(N / 32, M / 32);
    forward_kernel<<<blocks, threads>>>(input.const_data_ptr<float>(), output.mutable_data_ptr<float>(), M, N);
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) { m.def("forward", &forward); }
